#include "hip/hip_runtime.h"
#include <stdio.h>
#include <getopt.h>
#include <stdlib.h>
#include "CycleTimer.h"
#include "helper.h"
#include "bmpfuncs.h"
#include <hip/hip_runtime.h>
typedef unsigned char uchar;

void storeImage(float *imageOut, const char *filename, int rows, int cols,
                const char *refFilename)
{

   FILE *ifp, *ofp;
   unsigned char tmp;
   int offset;
   unsigned char *buffer;
   int i, j;

   int bytes;

   int height, width;

   ifp = fopen(refFilename, "rb");
   if (ifp == NULL)
   {
      perror(filename);
      exit(-1);
   }

   fseek(ifp, 10, SEEK_SET);
   fread(&offset, 4, 1, ifp);

   fseek(ifp, 18, SEEK_SET);
   fread(&width, 4, 1, ifp);
   fread(&height, 4, 1, ifp);

   fseek(ifp, 0, SEEK_SET);

   buffer = (unsigned char *)malloc(offset);
   if (buffer == NULL)
   {
      perror("malloc");
      exit(-1);
   }

   fread(buffer, 1, offset, ifp);

   printf("Writing output image to %s\n", filename);
   ofp = fopen(filename, "wb");
   if (ofp == NULL)
   {
      perror("opening output file");
      exit(-1);
   }
   bytes = fwrite(buffer, 1, offset, ofp);
   if (bytes != offset)
   {
      printf("error writing header!\n");
      exit(-1);
   }

   // NOTE bmp formats store data in reverse raster order (see comment in
   // readImage function), so we need to flip it upside down here.
   int mod = width % 4;
   if (mod != 0)
   {
      mod = 4 - mod;
   }
   //   printf("mod = %d\n", mod);
   for (i = height - 1; i >= 0; i--)
   {
      for (j = 0; j < width; j++)
      {
         tmp = (unsigned char)imageOut[i * cols + j];
         fwrite(&tmp, sizeof(char), 1, ofp);
      }
      // In bmp format, rows must be a multiple of 4-bytes.
      // So if we're not at a multiple of 4, add junk padding.
      for (j = 0; j < mod; j++)
      {
         fwrite(&tmp, sizeof(char), 1, ofp);
      }
   }

   fclose(ofp);
   fclose(ifp);

   free(buffer);
}

/*
 * Read bmp image and convert to byte array. Also output the width and height
 */
float *readImage(const char *filename, int *widthOut, int *heightOut)
{

   uchar *imageData;

   int height, width;
   uchar tmp;
   int offset;
   int i, j;

   printf("Reading input image from %s\n", filename);
   FILE *fp = fopen(filename, "rb");
   if (fp == NULL)
   {
      perror(filename);
      exit(-1);
   }

   fseek(fp, 10, SEEK_SET);
   fread(&offset, 4, 1, fp);

   fseek(fp, 18, SEEK_SET);
   fread(&width, 4, 1, fp);
   fread(&height, 4, 1, fp);

   printf("width = %d\n", width);
   printf("height = %d\n", height);

   *widthOut = width;
   *heightOut = height;

   imageData = (uchar *)malloc(width * height);
   if (imageData == NULL)
   {
      perror("malloc");
      exit(-1);
   }

   fseek(fp, offset, SEEK_SET);
   fflush(NULL);

   int mod = width % 4;
   if (mod != 0)
   {
      mod = 4 - mod;
   }

   // NOTE bitmaps are stored in upside-down raster order.  So we begin
   // reading from the bottom left pixel, then going from left-to-right,
   // read from the bottom to the top of the image.  For image analysis,
   // we want the image to be right-side up, so we'll modify it here.

   // First we read the image in upside-down

   // Read in the actual image
   for (i = 0; i < height; i++)
   {

      // add actual data to the image
      for (j = 0; j < width; j++)
      {
         fread(&tmp, sizeof(char), 1, fp);
         imageData[i * width + j] = tmp;
      }
      // For the bmp format, each row has to be a multiple of 4,
      // so I need to read in the junk data and throw it away
      for (j = 0; j < mod; j++)
      {
         fread(&tmp, sizeof(char), 1, fp);
      }
   }

   // Then we flip it over
   int flipRow;
   for (i = 0; i < height / 2; i++)
   {
      flipRow = height - (i + 1);
      for (j = 0; j < width; j++)
      {
         tmp = imageData[i * width + j];
         imageData[i * width + j] = imageData[flipRow * width + j];
         imageData[flipRow * width + j] = tmp;
      }
   }

   fclose(fp);

   // Input image on the host
   float *floatImage = NULL;
   floatImage = (float *)malloc(sizeof(float) * width * height);
   if (floatImage == NULL)
   {
      perror("malloc");
      exit(-1);
   }

   // Convert the BMP image to float (not required)
   for (i = 0; i < height; i++)
   {
      for (j = 0; j < width; j++)
      {
         floatImage[i * width + j] = (float)imageData[i * width + j];
      }
   }

   free(imageData);
   return floatImage;
}

//#include "serialConv.h"
char *readSource(char *kernelPath)
{
    cl_int status;
    FILE *fp;
    char *source;
    long int size;

    printf("Program file is: %s\n", kernelPath);

    fp = fopen(kernelPath, "rb");
    if (!fp)
    {
        printf("Could not open kernel file\n");
        exit(-1);
    }
    status = fseek(fp, 0, SEEK_END);
    if (status != 0)
    {
        printf("Error seeking to end of file\n");
        exit(-1);
    }
    size = ftell(fp);
    if (size < 0)
    {
        printf("Error getting file position\n");
        exit(-1);
    }

    rewind(fp);

    source = (char *)malloc(size + 1);

    int i;
    for (i = 0; i < size + 1; i++)
    {
        source[i] = '\0';
    }

    if (source == NULL)
    {
        printf("Error allocating space for the kernel source\n");
        exit(-1);
    }

    fread(source, 1, size, fp);
    source[size] = '\0';

    return source;
}
float *readFilter(const char *filename, int *filterWidth)
{
    printf("Reading filter data from %s\n", filename);

    FILE *fp = fopen(filename, "r");
    if (!fp)
    {
        printf("Could not open filter file\n");
        exit(-1);
    }

    fscanf(fp, "%d", filterWidth);

    float *filter = (float *)malloc(*filterWidth * *filterWidth * sizeof(int));

    float tmp;
    for (int i = 0; i < *filterWidth * *filterWidth; i++)
    {
        fscanf(fp, "%f", &tmp);
        filter[i] = tmp;
    }

    printf("Filter width: %d\n", *filterWidth);

    fclose(fp);
    return filter;
}

void usage(const char *progname)
{
   printf("Usage: %s [options]\n", progname);
   printf("Program Options:\n");
   printf("  -i  --input   <String> Input image\n");
   printf("  -f  --filter  <INT>    Use which filter (0, 1, 2)\n");
   printf("  -?  --help             This message\n");
}

int compare(const void *a, const void *b)
{
   double *x = (double *)a;
   double *y = (double *)b;
   if (*x < *y)
      return -1;
   else if (*x > *y)
      return 1;
   return 0;
}
__global__ void convolution(const float *img, float *out, const int width, 
                            const int height, float *filter, const int filterWidth)
{
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int col = index % width;
   int row = index / width;
   int half_filter = filterWidth / 2;
   float sum = 0;

   for(int k = -half_filter; k <= half_filter; k++) 
   {
        if(row + k < 0 || row + k >= height)
            continue;
        for(int l = -half_filter; l <= half_filter; l++)
        {
            
            if (col + l >= 0 && col + l < width)
            {
                sum += img[(row + k) * width + col + l] * filter[(k + half_filter) * filterWidth + l + half_filter];
            }
        }
   }
   
   out[row * width + col] = sum;
}


void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage)
{
    int filterSize = filterWidth * filterWidth;

    // Allocate device memory
    float *d_inputImage, *d_outputImage, *d_filter;
    hipMalloc((void**)&d_inputImage, imageHeight * imageWidth * sizeof(float));
    hipMalloc((void**)&d_outputImage, imageHeight * imageWidth * sizeof(float));
    hipMalloc((void**)&d_filter, filterSize * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_inputImage, inputImage, imageHeight * imageWidth * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, filter, filterSize * sizeof(float), hipMemcpyHostToDevice);

    // Set grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((imageWidth + blockSize.x - 1) / blockSize.x, (imageHeight + blockSize.y - 1) / blockSize.y);

    // Launch kernel
    convolution<<<gridSize, blockSize>>>(d_inputImage, d_outputImage, imageWidth, imageHeight, d_filter, filterWidth);

    // Copy output data from device to host
    hipMemcpy(outputImage, d_outputImage, imageHeight * imageWidth * sizeof(float), hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_inputImage);
    hipFree(d_outputImage);
    hipFree(d_filter);
}

int main(int argc, char **argv)
{
   int i, j;

   // Rows and columns in the input image
   int imageHeight;
   int imageWidth;

   double start_time, end_time;

   char *inputFile = "input.bmp";
   const char *outputFile = "output.bmp";
   //const char *refFile = "ref.bmp";
   char *filterFile = "filter1.csv";

   // parse commandline options ////////////////////////////////////////////
   int opt;
   static struct option long_options[] = {
       {"filter", 1, 0, 'f'},
       {"input", 1, 0, 'i'},
       {"help", 0, 0, '?'},
       {0, 0, 0, 0}};

   while ((opt = getopt_long(argc, argv, "i:f:?", long_options, NULL)) != EOF)
   {

      switch (opt)
      {
      case 'i':
      {
         inputFile = optarg;

         break;
      }
      case 'f':
      {
         int idx = atoi(optarg);
         if (idx == 2)
            filterFile = "filter2.csv";
         else if (idx == 3)
            filterFile = "filter3.csv";

         break;
      }
      case '?':
      default:
         usage(argv[0]);
         return 1;
      }
   }
   // end parsing of commandline options

   // read filter data
   int filterWidth;
   float *filter = readFilter(filterFile, &filterWidth);

   // Homegrown function to read a BMP from file
   float *inputImage = readImage(inputFile, &imageWidth, &imageHeight);
   // Size of the input and output images on the host
   int dataSize = imageHeight * imageWidth * sizeof(float);
   // Output image on the host
   float *outputImage = (float *)malloc(dataSize);

   // helper init CL
   //cl_program program;
   //cl_device_id device;
   //cl_context context;
   //initCL(&device, &context, &program);

   double minThread = 0;
   double recordThread[10] = {0};
   for (int i = 0; i < 10; ++i)
   {
      memset(outputImage, 0, dataSize);
      start_time = currentSeconds();
      // Run the host to execute the kernel
      hostFE(filterWidth, filter, imageHeight, imageWidth, inputImage, outputImage);
      end_time = currentSeconds();
      recordThread[i] = end_time - start_time;
   }
   qsort(recordThread, 10, sizeof(double), compare);
   for (int i = 3; i < 7; ++i)
   {
      minThread += recordThread[i];
   }
   minThread /= 4;

   printf("\n[conv opencl]:\t\t[%.3f] ms\n\n", minThread * 1000);

   // Write the output image to file
   storeImage(outputImage, outputFile, imageHeight, imageWidth, inputFile);

   // Output image of reference on the host
//    float *refImage = NULL;
//    refImage = (float *)malloc(dataSize);
//    memset(refImage, 0, dataSize);

//    double minSerial = 0;
//    double recordSerial[10] = {0};
//    for (int i = 0; i < 10; ++i)
//    {
//       memset(refImage, 0, dataSize);
//       start_time = currentSeconds();
//       serialConv(filterWidth, filter, imageHeight, imageWidth, inputImage, refImage);
//       end_time = currentSeconds();
//       recordSerial[i] = end_time - start_time;
//    }
//    qsort(recordSerial, 10, sizeof(double), compare);
//    for (int i = 3; i < 7; ++i)
//    {
//       minSerial += recordSerial[i];
//    }
//    minSerial /= 4;

//    printf("\n[conv serial]:\t\t[%.3f] ms\n\n", minSerial * 1000);

//    storeImage(refImage, refFile, imageHeight, imageWidth, inputFile);

//    int diff_counter = 0;
//    for (i = 0; i < imageHeight; i++)
//    {
//       for (j = 0; j < imageWidth; j++)
//       {
//          if (abs(outputImage[i * imageWidth + j] - refImage[i * imageWidth + j]) > 10)
//          {
//             diff_counter += 1;
//          }
//       }
//    }

//    float diff_ratio = (float)diff_counter / (imageHeight * imageWidth);
//    printf("Diff ratio: %f\n", diff_ratio);

//    if (diff_ratio > 0.1)
//    {
//       printf("\n\033[31mFAILED:\tResults are incorrect!\033[0m\n");
//       return -1;
//    }
//    else
//    {
//       printf("\n\033[32mPASS:\t(%.2fx speedup over the serial version)\033[0m\n", minSerial / minThread);
//    }

   return 0;
}
