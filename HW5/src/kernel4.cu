//---3.792ms---
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, int resX, int resY, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX < resX && thisY < resY){

		float x = lowerX + thisX * stepX;
		float y = lowerY + thisY * stepY;
		int index = thisY * resX + thisX;
		d_img[index] = mandel(x, y, maxIterations);
	}
    
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{   
    
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    //initiate the memory
    dim3 threadsofblock(16,16);
    dim3 totalblocks(ceil(resX / threadsofblock.x), ceil(resY / threadsofblock.y));

    //not allowed to use the image input as the host memory directly
    int size = resX * resY * sizeof(int);
    int *d_img;
    //int *host = (int*)malloc(size);
    hipMalloc((void**)&d_img, size);

    //launch the kernel
    mandelKernel<<<totalblocks, threadsofblock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations);

    hipMemcpy(img, d_img, size, hipMemcpyDeviceToHost);
    //memcpy(img, host, size);

    //free the memory
    hipFree(d_img);
    //free(host);
    
}