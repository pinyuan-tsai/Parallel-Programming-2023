//kernel 0
// #include <cuda.h>
// #include <stdio.h>
// #include <stdlib.h>

// __global__ void mandelKernel() {
//     // To avoid error caused by the floating number, use the following pseudo code
//     //
//     // float x = lowerX + thisX * stepX;
//     // float y = lowerY + thisY * stepY;
// }

// // Host front-end function that allocates the memory and launches the GPU kernel
// void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
// {
//     float stepX = (upperX - lowerX) / resX;
//     float stepY = (upperY - lowerY) / resY;
// }

//kernel 1
// #include <cuda.h>
// #include <stdio.h>
// #include <stdlib.h>

// __device__ int mandel(float c_re, float c_im, int maxIteration)
// {
//   float z_re = c_re, z_im = c_im;
//   int i;
//   for (i = 0; i < maxIteration; ++i)
//   {

//     if (z_re * z_re + z_im * z_im > 4.f)
//       break;

//     float new_re = z_re * z_re - z_im * z_im;
//     float new_im = 2.f * z_re * z_im;
//     z_re = c_re + new_re;
//     z_im = c_im + new_im;
//   }

//   return i;
// }

// __global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, int resX, int resY, int maxIterations){
//     // To avoid error caused by the floating number, use the following pseudo code
//     //
//     // float x = lowerX + thisX * stepX;
//     // float y = lowerY + thisY * stepY;

//     int thisX = blockIdx.x * blockDim.x + threadIdx.x;
//     int thisY = blockIdx.y * blockDim.y + threadIdx.y;

//     if (thisX >= resX || thisY >= resY) return;

//     float x = lowerX + thisX * stepX;
//     float y = lowerY + thisY * stepY;
//     int index = thisY * resX + thisX;
//     d_img[index] = mandel(x, y, maxIterations);
    
// }

// // Host front-end function that allocates the memory and launches the GPU kernel
// void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
// {   
    
//     float stepX = (upperX - lowerX) / resX;
//     float stepY = (upperY - lowerY) / resY;

//     //initiate the memory
//     dim3 threadsofblock(16,16);
//     dim3 totalblocks(ceil(resX / threadsofblock.x), ceil(resY / threadsofblock.y));

//     //not allowed to use the image input as the host memory directly
//     int size = resX * resY * sizeof(int);
//     int *d_img;
//     int *host = (int*)malloc(size);
//     cudaMalloc((void**)&d_img, size);

//     //launch the kernel
//     mandelKernel<<<totalblocks, threadsofblock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations);

//     cudaMemcpy(host, d_img, size, cudaMemcpyDeviceToHost);
//     memcpy(img, host, size);

//     //free the memory
//     cudaFree(d_img);
//     free(host);
    
// }

//kernel2
// #include <cuda.h>
// #include <stdio.h>
// #include <stdlib.h>

// __device__ int mandel(float c_re, float c_im, int maxIteration)
// {
//   float z_re = c_re, z_im = c_im;
//   int i;
//   for (i = 0; i < maxIteration; ++i)
//   {

//     if (z_re * z_re + z_im * z_im > 4.f)
//       break;

//     float new_re = z_re * z_re - z_im * z_im;
//     float new_im = 2.f * z_re * z_im;
//     z_re = c_re + new_re;
//     z_im = c_im + new_im;
//   }

//   return i;
// }

// __global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, int resX, int resY, int maxIterations){
//     // To avoid error caused by the floating number, use the following pseudo code
//     //
//     // float x = lowerX + thisX * stepX;
//     // float y = lowerY + thisY * stepY;

//     int thisX = blockIdx.x * blockDim.x + threadIdx.x;
//     int thisY = blockIdx.y * blockDim.y + threadIdx.y;

//     if (thisX >= resX || thisY >= resY) return;

//     float x = lowerX + thisX * stepX;
//     float y = lowerY + thisY * stepY;
//     int index = thisY * resX + thisX;
//     d_img[index] = mandel(x, y, maxIterations);
    
// }

// // Host front-end function that allocates the memory and launches the GPU kernel
// void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
// {   
    
//     float stepX = (upperX - lowerX) / resX;
//     float stepY = (upperY - lowerY) / resY;

//     //initiate the memory
//     dim3 threadsofblock(16,16);
//     dim3 totalblocks(ceil(resX / threadsofblock.x), ceil(resY / threadsofblock.y));

//     //Use cudaHostAlloc to allocate the host memory, and use cudaMallocPitch to allocate GPU memory
//     int size = resX * resY * sizeof(int);
//     int *d_img;
//     int *host;
//     size_t pitch = 0;
//     // Allocate pinned host memory using cudaHostAlloc
//     cudaHostAlloc(&host, size, cudaHostAllocDefault);
//     // Allocate device memory using cudaMallocPitch
//     cudaMallocPitch(&d_img, &pitch, resX * sizeof(int), resY);

//     //launch the kernel
//     mandelKernel<<<totalblocks, threadsofblock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations);

//     cudaDeviceSynchronize();
    
//     //copy the memory back
//     //printf("%d\n", pitch);
//     //cudaMemcpy2D(host, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, cudaMemcpyDeviceToHost);
//     cudaMemcpy(host,d_img, size, cudaMemcpyDeviceToHost);
//     memcpy(img, host, size);

//     //free the memory
//     cudaFree(d_img);
//     cudaFreeHost(host);
    
// }


//kernel3
//---5.794ms---
// #include <cuda.h>
// #include <stdio.h>
// #include <stdlib.h>

// __device__ int mandel(float c_re, float c_im, int maxIteration)
// {
//   float z_re = c_re, z_im = c_im;
//   int i;
//   for (i = 0; i < maxIteration; ++i)
//   {

//     if (z_re * z_re + z_im * z_im > 4.f)
//       break;

//     float new_re = z_re * z_re - z_im * z_im;
//     float new_im = 2.f * z_re * z_im;
//     z_re = c_re + new_re;
//     z_im = c_im + new_im;
//   }

//   return i;
// }

// __global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, int resX, int resY, int maxIterations, int group_x, int group_y){
//     // To avoid error caused by the floating number, use the following pseudo code
//     //
//     // float x = lowerX + thisX * stepX;
//     // float y = lowerY + thisY * stepY;

//     int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * group_x;
//     int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * group_y;

//     if (thisX >= resX || thisY >= resY) return;

//     for(int i = 0; i < group_y; i++){
//       if(thisY + i >= resY) return;
//       for(int j = 0; j < group_x ; j++){
//         if(thisX + j >= resX) return;
//         float x = lowerX + (thisX + j) * stepX;
//         float y = lowerY + (thisY + i) * stepY;
//         int index = (thisY + i) * resX + (thisX + j);
//         d_img[index] = mandel(x, y, maxIterations);
//       }
//     }

//     // float x = lowerX + thisX * stepX;
//     // float y = lowerY + thisY * stepY;
//     // int index = thisY * resX + thisX;
//     // d_img[index] = mandel(x, y, maxIterations);
    
// }

// // Host front-end function that allocates the memory and launches the GPU kernel
// void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
// {   
    
//     float stepX = (upperX - lowerX) / resX;
//     float stepY = (upperY - lowerY) / resY;
//     int group_x = 2, group_y = 2;
//     int thread = 16;
//     //initiate the memory
//     dim3 threadsofblock(thread / group_x ,thread / group_y);
//     dim3 totalblocks(ceil(resX / thread), ceil(resY / thread));

//     //Use cudaHostAlloc to allocate the host memory, and use cudaMallocPitch to allocate GPU memory
//     int size = resX * resY * sizeof(int);
//     int *d_img;
//     int *host;
//     size_t pitch = 0;
//     // Allocate pinned host memory using cudaHostAlloc
//     cudaHostAlloc(&host, size, cudaHostAllocDefault);
//     // Allocate device memory using cudaMallocPitch
//     cudaMallocPitch(&d_img, &pitch, resX * sizeof(int), resY);

//     //launch the kernel
//     mandelKernel<<<totalblocks, threadsofblock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations, group_x,group_y);

//     cudaDeviceSynchronize();
    
//     //copy the memory back
//     //printf("%d\n", pitch);
//     //cudaMemcpy2D(host, resX * sizeof(int), d_img, pitch, resX * sizeof(int), resY, cudaMemcpyDeviceToHost);
//     cudaMemcpy(host,d_img, size, cudaMemcpyDeviceToHost);
//     memcpy(img, host, size);

//     //free the memory
//     cudaFree(d_img);
//     cudaFreeHost(host);
    
// }

//kernel4
//---3.792ms---
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int maxIteration)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < maxIteration; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int *d_img, int resX, int resY, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    if (thisX < resX && thisY < resY){

		float x = lowerX + thisX * stepX;
		float y = lowerY + thisY * stepY;
		int index = thisY * resX + thisX;
		d_img[index] = mandel(x, y, maxIterations);
	}
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{   
    
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    //initiate the memory
    dim3 threadsofblock(16,16);
    dim3 totalblocks(ceil(resX / threadsofblock.x), ceil(resY / threadsofblock.y));

    //not allowed to use the image input as the host memory directly
    int size = resX * resY * sizeof(int);
    int *d_img;
    //int *host = (int*)malloc(size);
    hipMalloc((void**)&d_img, size);

    //launch the kernel
    mandelKernel<<<totalblocks, threadsofblock>>>(lowerX, lowerY, stepX, stepY, d_img, resX, resY, maxIterations);

    hipMemcpy(img, d_img, size, hipMemcpyDeviceToHost);
    //memcpy(img, host, size);

    //free the memory
    hipFree(d_img);
    //free(host);
    
}